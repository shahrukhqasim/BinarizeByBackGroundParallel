#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_code.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <algorithm>
#include <stdio.h>
#include <math_functions.h>

__global__ void bg_estimate(uchar*data,uchar*data_out, int rows, int cols, int rRows, int rCols, int numPixels) {
    __shared__ uchar sharedMemory[4096];

    int pixelLocation = blockIdx.x*1024+threadIdx.x;
    if(pixelLocation >= numPixels)
        return;
    data_out[pixelLocation] = 37;

    int posX = pixelLocation%rCols;
    int posY = pixelLocation/rCols;

    if (posX < WINDOW_SIZE || posY < WINDOW_SIZE )
        return;
    if ((posX > (WINDOW_SIZE + cols)) || (posY > (WINDOW_SIZE + rows)))
        return;



    __syncthreads();

    unsigned short histogram[256] = {};

    for (int i = posY - WINDOW_SIZE ; i <= posY + WINDOW_SIZE ; i++) {
        for (int j = posX - WINDOW_SIZE ; j <= posX + WINDOW_SIZE ; j++) {
//            histogram[data[i*rCols + j]]++;
            histogram[0]++;
        }
    }

    __syncthreads();

    int counter = 0;
    int limit = (int) (0.8 * (WINDOW_SIZE+1)*(WINDOW_SIZE+1));
    int mean = 0;

    for (int i = 255; counter < limit; i--) {
        mean += histogram[i] * i;
        counter += histogram[i];
    }
    mean /= counter < 1 ? 1 : counter;

//    data[pixelLocation] = 1*255;

    data_out[pixelLocation] = 37;//255.0 * min(1.0, double(data[pixelLocation]) / mean);
}

__global__ void bg_estimate2(uchar*data,uchar*data_out, int rows, int cols, int rRows, int rCols, int numPixels) {
    __shared__ uchar sharedMemory[4096];

}

void cuda_binarize(uchar*data, uchar*data2, int rows, int cols, int rRows, int rCols) {
    printf("Hello from cu file\n");

    uchar*d_data;
    uchar*d_data_out;

    const dim3 blockSize(1024,1,1);

    int N = rRows*rCols;
    const dim3 gridSize(N/1024+N%1024,1,1);

    hipMalloc(&d_data, sizeof(uchar) * rRows * rCols);
    hipMalloc(&d_data_out, sizeof(uchar) * rRows * rCols);
    hipMemcpy(d_data, data , sizeof(uchar) * rRows * rCols ,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("%d", rCols);
    bg_estimate2 <<<gridSize,blockSize, 4906 * sizeof(uchar)>>>(d_data, d_data_out, rows, cols, rRows, rCols, rRows*rCols);
    hipDeviceSynchronize();
    hipMemcpy(data, d_data_out , sizeof(uchar) * rRows * rCols ,hipMemcpyDeviceToHost);
    printf("Second hello 2");
}